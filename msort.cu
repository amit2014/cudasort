#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

#include "sort.h"
#include "common.h"

__device__ void mSort_helper(dataType *data, int n, dataType *res)   {
    // printf("Thread %d\n", omp_get_thread_num());
    if(n == 1)  {
        res[0] = data[0];
        return;
    }
    if(n <= 0) {
        return;
    }
    
    mSort_helper(res, n/2, data);
 
    mSort_helper(res+n/2, n-n/2, data+n/2);

    merge(data, n/2, n, res);
}

__global__ void mSortKernel(dataType *data, int n, dataType *res)  {
    bottomUpMergeSort(data, n, res);
    // mSort_helper(res, n, data);
}

void mSort(dataType *data, int n)    {
    dataType *buf1, *buf2;
    hipMalloc((void**)&buf1, n*sizeof(dataType));
    hipMalloc((void**)&buf2, n*sizeof(dataType));

    hipMemcpy(buf1, data, n*sizeof(dataType), hipMemcpyHostToDevice);
    hipMemcpy(buf2, data, n*sizeof(dataType), hipMemcpyHostToDevice);
    //printf("%d threads\n", omp_get_num_threads());
    mSortKernel <<< 1,2 >>> (buf1, n, buf2);

    hipMemcpy(data, buf1, n*sizeof(dataType), hipMemcpyDeviceToHost);
    hipFree(buf1);
    hipFree(buf2);
}

